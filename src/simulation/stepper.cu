#include "hip/hip_runtime.h"
//
// Created by jay on 9/30/24.
//

#include <stdexcept>

#include "stepper.cuh"

using namespace cu_sim;

__global__ void setup_kernel(
  const body *__restrict__ bodies, float3 *__restrict__ history, float3 *__restrict__ line_color,
  const size_t count, const size_t hist_size
) {
  const auto idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(idx >= count) return;
  for(size_t  i = 0; i < hist_size; i++)
    history[hist_size * idx + i] = float3(bodies[idx].position.x, bodies[idx].position.y, bodies[idx].position.z);
  line_color[idx] = float3(bodies[idx].color.x, bodies[idx].color.y, bodies[idx].color.z);
}

#include <iostream>
stepper::stepper(const buffer_handles &buffers, const line_handles &handles, const std::vector<body> &initial_state, const size_t hist_size, const size_t hist_skip)
  : pos_buf{buffers.pos}, radius_buf{buffers.radius}, color_buf{buffers.color}, history_buf{handles.history}, line_color_buf{handles.color},
    history_length{hist_size}, history_skip{hist_skip}
{
  const size_t count = initial_state.size();
  if(pos_buf.element_count() != count) {
    throw std::runtime_error("Mismatched buffer sizes: " + std::to_string(count) + " != " + std::to_string(pos_buf.element_count()));
  }
  if(radius_buf.element_count() != count) {
    throw std::runtime_error("Mismatched buffer sizes: " + std::to_string(count) + " != " + std::to_string(radius_buf.element_count()));
  }
  if(color_buf.element_count() != count) {
    throw std::runtime_error("Mismatched buffer sizes: " + std::to_string(count) + " != " + std::to_string(color_buf.element_count()));
  }

  cuda_checked(hipMalloc(&bodies, count * sizeof(body)));
  cuda_checked(hipMemcpy(bodies, initial_state.data(), count * sizeof(body), hipMemcpyHostToDevice));
  cuda_checked(hipMalloc(&back_buffer, count * sizeof(body)));
  cuda_checked(hipMemcpy(back_buffer, initial_state.data(), count * sizeof(body), hipMemcpyHostToDevice));

  hipDeviceProp_t prop{};
  cuda_checked(hipGetDeviceProperties(&prop, 0));
  grid = dim3{static_cast<unsigned int>(count / prop.maxThreadsPerBlock + 1), 1, 1};
  block = dim3{static_cast<unsigned int>(prop.maxThreadsPerBlock), 1, 1};

  setup_kernel<<<grid, block>>>(bodies, history_buf, line_color_buf, count, history_length);
  cuda_checked(hipPeekAtLastError());
  cuda_checked(hipDeviceSynchronize());
}

__global__ void step_kernel(
  const body *__restrict__ bodies, body *__restrict__ back,
  const size_t count, const float dt
) {
  const auto idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(idx >= count) return;

  vec force{0, 0, 0};
  for(size_t i = 0; i < count; ++i) {
    if(idx == i) continue;
    vec direction = bodies[i].position - bodies[idx].position;
    const float distance = max(direction.length(), 1e-3f);
    direction = direction.normalized();
    force += direction * G * bodies[i].mass / (distance * distance); // F = G * m1 * m2 / r^2
  }

  // a = F / m
  const auto acceleration = force / bodies[idx].mass;

  // v += a * dt
  back[idx].velocity = bodies[idx].velocity + acceleration * dt;

  // x += v * dt
  back[idx].position = bodies[idx].position + back[idx].velocity * dt;
}

__global__ void copy_kernel(
  body *__restrict__ bodies, const body *__restrict__ back,
  float3 *__restrict__ pos, float *__restrict__ radius, float3 *__restrict__ color,
  float3 *__restrict__ history, float3 *__restrict__ line_color,
  const size_t count, const size_t history_size, const bool step_history
) {
  const auto idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(idx >= count) return;

  bodies[idx] = back[idx];

  pos[idx].x = bodies[idx].position.x;
  pos[idx].y = bodies[idx].position.y;
  pos[idx].z = bodies[idx].position.z;

  radius[idx] = bodies[idx].radius;

  color[idx].x = bodies[idx].color.x;
  color[idx].y = bodies[idx].color.y;
  color[idx].z = bodies[idx].color.z;

  line_color[idx] = color[idx];
  if(step_history) {
    for(size_t i = history_size - 1; i > 0; i--) {
      history[idx * history_size + i] = history[idx * history_size + i - 1];
    }
  }
  history[idx * history_size] = pos[idx];
}

__global__ void copy_kernel_no_history(
  body *__restrict__ bodies, const body *__restrict__ back,
  float3 *__restrict__ pos, float *__restrict__ radius, float3 *__restrict__ color,
  const size_t count
) {
  const auto idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(idx >= count) return;

  bodies[idx] = back[idx];

  pos[idx].x = bodies[idx].position.x;
  pos[idx].y = bodies[idx].position.y;
  pos[idx].z = bodies[idx].position.z;

  radius[idx] = bodies[idx].radius;

  color[idx].x = bodies[idx].color.x;
  color[idx].y = bodies[idx].color.y;
  color[idx].z = bodies[idx].color.z;
}

void stepper::step(const float dt, const size_t frame_idx) {
  step_kernel<<<grid, block>>>(bodies, back_buffer, pos_buf.element_count(), dt);
  cuda_checked(hipPeekAtLastError());
  cuda_checked(hipDeviceSynchronize());
  // explicit synchronization - forces all threads to complete before copying
  copy_kernel<<<grid, block>>>(
    bodies, back_buffer, pos_buf, radius_buf, color_buf, history_buf, line_color_buf, pos_buf.element_count(),
    history_length, (frame_idx % history_skip) == 0
  );
  cuda_checked(hipPeekAtLastError());
  cuda_checked(hipDeviceSynchronize());
}

void stepper::step_no_history(const float dt, const size_t frame_idx) {
  step_kernel<<<grid, block>>>(bodies, back_buffer, pos_buf.element_count(), dt);
  cuda_checked(hipPeekAtLastError());
  cuda_checked(hipDeviceSynchronize());
  // explicit synchronization - forces all threads to complete before copying
  copy_kernel_no_history<<<grid, block>>>(
    bodies, back_buffer, pos_buf, radius_buf, color_buf, pos_buf.element_count()
  );
  cuda_checked(hipPeekAtLastError());
  cuda_checked(hipDeviceSynchronize());
}

stepper::~stepper() {
  hipFree(bodies);
}