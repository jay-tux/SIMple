#include "hip/hip_runtime.h"
#include <iostream>

#include "rendering/shader.cuh"
#include "rendering/camera.cuh"
#include "rendering/object.cuh"
#include "rendering/gl_wrapper.cuh"
#include "cuda_gl_bridge.cuh"
#include "simulation/stepper.cuh"
#include "loader.cuh"

__global__ void micro_kernel(float3 *pos, float *radius, float3 *color, const size_t count, const float time) {
  auto idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(idx >= count) return;

  if(idx % 2 == 0) {
    pos[idx].x = sinf(time);
    pos[idx].y = 0;
    pos[idx].z = 0;

    radius[idx] = abs(cosf(time));

    color[idx].x = 1.0f - abs(tanhf(sinf(time)));
    color[idx].y = abs(tanhf(cosf(time)));
    color[idx].z = 0.0f;
  }
  else {
    pos[idx].x = 0;
    pos[idx].y = cosf(time);
    pos[idx].z = 0;

    radius[idx] = abs(sinf(time));

    color[idx].x = 0.0f;
    color[idx].y = abs(tanhf(sinf(time)));
    color[idx].z = 1.0f - abs(tanhf(cosf(time)));
  }
}

constexpr static float time_scale = 2e2f;

int main(const int argc, const char **argv) {
  if(argc != 2) {
    std::cerr << "Usage: " << argv[0] << " <path to INI>\n"
              << "\n"
              << " -> INI spec:\n"
              << "    -> 1 section ([object name]) per body in the simulation\n"
              << "    -> Each section must have the following keys:\n"
              << "       -> position: vec3 (three floating-point literals, separated by spaces)\n"
              << "       -> velocity: vec3\n"
              << "       -> mass: float\n"
              << "    -> Optional keys per section:\n"
              << "       -> radius: float (default: 0.1); specifies the rendering radius\n"
              << "       -> color: vec3 (default: 0.22 0.22 0.22); specifies the color of the body in the rendering\n"
              << "       -> mass_div_g: bool (default: false); specifies whether or not to divide the mass by G\n"
              << "       -> pos_div_g: bool (default: false); specifies whether or not to divide the position by G\n"
              << "       -> vel_div_g: bool (default: false); specifies whether or not to divide the velocity by G\n"
              << "       -> radius_div_g: bool (default: false); specifies whether or not to divide the radius by G\n"
              << "    -> The G value used is " << cu_sim::G << "\n"
              << "    -> All other keys are ignored.\n";
    return -1;
  }

  try {
    const auto bodies = cu_sim::load_bodies(argv[1]);
    const cu_sim::shader shader("shaders/vertex.glsl", "shaders/fragment.glsl");
    const cu_sim::object object("assets/body.obj", bodies.size());
    cu_sim::camera camera({0, 0, 7}, {0, 0, 0}, {0, 1, 0});
    camera.fov_y = 90.0f;

    cu_sim::stepper simulator(object.cuda_buffers(), bodies);

    auto &render = cu_sim::gl_wrapper::get();
    render.clear();
    render.frame();
    bool step = true;
    while(!render.should_close()) {
      render.clear();
      if(render.toggle_progress()) step = !step;

      if(step)
        simulator.step(render.delta_time() * time_scale);

      shader.enable();
      shader.set_m4(1, camera.view_matrix());
      shader.set_m4(2, camera.projection_matrix());
      object.draw();

      render.frame();
    }
  }
  catch(const std::exception &e) {
    std::cerr << e.what() << "\n";
    return 1;
  }
}
